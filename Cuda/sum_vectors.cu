
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <string>
#include <cstdlib>

using namespace std;

#define N 10

__global__ void  addKernel(int *a, int *b, int *c)
{
	int tid = blockIdx.x;

	if ( tid < N )
	{
		c[tid] = a[tid] + b[tid];
	}
}

int main()
{
	int *a, *b, *c;

	a = (int*) malloc(sizeof(int) * N);
	b = (int*) malloc(sizeof(int) * N);
	c = (int*) malloc(sizeof(int) * N);

	int *dev_a, *dev_b, *dev_c;

	
	if ( hipErrorOutOfMemory == hipMalloc( (void**)&dev_a, N * sizeof(int) ) )
	{
		fprintf(stderr, "Error allocating memory\n");
		return -1;
	}
	if ( hipErrorOutOfMemory == hipMalloc( (void**)&dev_b, N * sizeof(int) ) )
	{
		fprintf(stderr, "Error allocating memory\n");
		return -1;
	}
	if ( hipErrorOutOfMemory == hipMalloc( (void**)&dev_c, N * sizeof(int) ) )
	{
		fprintf(stderr, "Error allocating memory\n");
		return -1;
	}
	

	for (int i=0; i < N; ++i)
	{
		a[i]= i;
		b[i]= i;
	}

	int retA = hipMemcpy( dev_a, a, N*sizeof(int), hipMemcpyDefault );

	if (retA == hipSuccess) {
		printf("A is Ok!\n");
	} 
	else if ( retA == hipErrorInvalidValue )
	{
		fprintf(stderr, "Invalid value\n");
		return -1;
	}
	else if ( retA == hipErrorInvalidDevicePointer )
	{
		fprintf(stderr, "Invalid pointer\n");
		return -1;
	}
	else
	{
		fprintf(stderr, "Invalid direction\n");
		printf("Error type %x\n",retA );
		return -1;
	}


	if ( hipMemcpy( dev_b, b, N*sizeof(int), hipMemcpyHostToDevice ) != hipSuccess )
	{
		fprintf(stderr, "Something goes wrong in dev_b\n");
	}

	addKernel<<<N,1>>>(dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost );

	for (int i=0;i<N;++i)
	{
		printf("%d + %d = %d\n",a[i], b[i], c[i]);
	}
	printf("\n");

	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );

	return 0;
}